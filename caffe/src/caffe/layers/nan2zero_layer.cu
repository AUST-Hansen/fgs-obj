#include "hip/hip_runtime.h"
/*
 * nan2zero_layer.cpp
 *
 *  Created on: May 17, 2014
 *      Author: zhangyuting
 */


#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void NaN2ZeroForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = (in[index] == in[index]) ? in[index] : 0;
  }
}

template <typename Dtype>
Dtype NaN2ZeroLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  NaN2ZeroForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  return Dtype(0);
}

template <typename Dtype>
__global__ void NaN2ZeroBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = (in_diff[index] == in_diff[index] &&
    		in_data[index] == in_data[index])? in_diff[index] : 0;
    // the second condition, in_data[index] == in_data[index], should not be useful,
    // as in_data must be non-nan after feedforward
  }
}

template <typename Dtype>
void NaN2ZeroLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    NaN2ZeroBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_CLASS(NaN2ZeroLayer);


}  // namespace caffe

